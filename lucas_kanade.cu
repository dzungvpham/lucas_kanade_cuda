#include "hip/hip_runtime.h"
/**
 * (c) 2019 Dzung Pham
 */

#include <math.h>
#include <hip/hip_math_constants.h> // CUDA Math constants
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "jpeglib.h"

// ----------------------- Helper code ------------------------

/*
 * IMAGE DATA FORMATS:
 *
 * The standard input image format is a rectangular array of pixels, with
 * each pixel having the same number of "component" values (color channels).
 * Each pixel row is an array of JSAMPLEs (which typically are unsigned chars).
 * If you are working with color data, then the color values for each pixel
 * must be adjacent in the row; for example, R,G,B,R,G,B,R,G,B,... for 24-bit
 * RGB color.
 */

/* The "frame structure" structure contains an image frame (in RGB or grayscale
 * formats) for passing around the CS338 projects.
 */
typedef struct frame_struct {
    JSAMPLE *image_buffer;	/* Points to large array of R,G,B-order/grayscale data
                             * Access directly with:
                             *   image_buffer[num_channels*pixel + component]
                             */
    JSAMPLE **row_pointers;	/* Points to an array of pointers to the beginning
                             * of each row in the image buffer.  Use to access
                             * the image buffer in a row-wise fashion, with:
                             *   row_pointers[row][num_channels*pixel + component]
                             */
    int image_height;		/* Number of rows in image */
    int image_width;		/* Number of columns in image */
    int num_channels;	/* Number of components (usually RGB=3 or gray=1) */
} frame_struct_t;
typedef frame_struct_t *frame_ptr;

/* Read/write JPEGs, for program startup & shutdown */
void write_JPEG_file (const char * filename, frame_ptr p_info, int quality);
frame_ptr read_JPEG_file (char * filename);

/* Allocate/deallocate frame buffers */
frame_ptr allocate_frame(int height, int width, int num_channels);
void destroy_frame(frame_ptr kill_me);

/*
 * write_JPEG_file writes out the contents of an image buffer to a JPEG.
 * A quality level of 2-100 can be provided (default = 75, high quality = ~95,
 * low quality = ~25, utter pixellation = 2).  Note that unlike read_JPEG_file,
 * it does not do any memory allocation on the buffer passed to it.
 */
void write_JPEG_file (const char * filename, frame_ptr p_info, int quality) {
    struct jpeg_compress_struct cinfo;
    struct jpeg_error_mgr jerr;
    FILE * outfile;		/* target file */

    /* Step 1: allocate and initialize JPEG compression object */
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_compress(&cinfo);

    /* Step 2: specify data destination (eg, a file) */
    /* Note: steps 2 and 3 can be done in either order. */

    if ((outfile = fopen(filename, "wb")) == NULL) {
        fprintf(stderr, "ERROR: Can't open output file %s\n", filename);
        exit(EXIT_FAILURE);
    }
    jpeg_stdio_dest(&cinfo, outfile);

    /* Step 3: set parameters for compression */

    /* Set basic picture parameters (not optional) */
    cinfo.image_width = p_info->image_width; 	/* image width and height, in pixels */
    cinfo.image_height = p_info->image_height;
    cinfo.input_components = p_info->num_channels; /* # of color components per pixel */
    if (p_info->num_channels == 3)
        cinfo.in_color_space = JCS_RGB; 	/* colorspace of input image */
    else if (p_info->num_channels == 1)
        cinfo.in_color_space = JCS_GRAYSCALE;
    else {
        fprintf(stderr, "ERROR: Non-standard colorspace for compressing!\n");
        exit(EXIT_FAILURE);
    }
    /* Fill in the defaults for everything else, then override quality */
    jpeg_set_defaults(&cinfo);
    jpeg_set_quality(&cinfo, quality, TRUE /* limit to baseline-JPEG values */);

    /* Step 4: Start compressor */
    jpeg_start_compress(&cinfo, TRUE);

    /* Step 5: while (scan lines remain to be written) */
    /*           jpeg_write_scanlines(...); */
    while (cinfo.next_scanline < cinfo.image_height) {
    (void) jpeg_write_scanlines(&cinfo, &(p_info->row_pointers[cinfo.next_scanline]), 1);
    }

    /* Step 6: Finish compression & close output */

    jpeg_finish_compress(&cinfo);
    fclose(outfile);

    /* Step 7: release JPEG compression object */
    jpeg_destroy_compress(&cinfo);
}

/*
 * read_JPEG_file reads the contents of a JPEG into an image buffer, which
 * is automatically allocated after the size of the image is determined.
 * We want to return a frame struct on success, NULL on error.
 */

frame_ptr read_JPEG_file (char * filename) {
    /* This struct contains the JPEG decompression parameters and pointers to
    * working space (which is allocated as needed by the JPEG library).
    */
    struct jpeg_decompress_struct cinfo;
    struct jpeg_error_mgr jerr;
    FILE * infile;		/* source file */
    frame_ptr p_info;		/* Output frame information */

    //  JSAMPLE *realBuffer;
    //  JSAMPLE **buffer;		/* Output row buffer */
    //  int row_stride;		/* physical row width in output buffer */

    /* Step 1: allocate and initialize JPEG decompression object */
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_decompress(&cinfo);

    /* Step 2: open & specify data source (eg, a file) */
    if ((infile = fopen(filename, "rb")) == NULL) {
        fprintf(stderr, "ERROR: Can't open input file %s\n", filename);
        exit(EXIT_FAILURE);
    }
    jpeg_stdio_src(&cinfo, infile);

    /* Step 3: read file parameters with jpeg_read_header() */
    (void) jpeg_read_header(&cinfo, TRUE);

    /* Step 4: use default parameters for decompression */

    /* Step 5: Start decompressor */
    (void) jpeg_start_decompress(&cinfo);

    /* Step X: Create a frame struct & buffers and fill in the blanks */
    fprintf(stderr, "  Opened %s: height = %d, width = %d, c = %d\n",
        filename, cinfo.output_height, cinfo.output_width, cinfo.output_components);
    p_info = allocate_frame(cinfo.output_height, cinfo.output_width, cinfo.output_components);

    /* Step 6: while (scan lines remain to be read) */
    /*           jpeg_read_scanlines(...); */
    while (cinfo.output_scanline < cinfo.output_height) {
        (void) jpeg_read_scanlines(&cinfo, &(p_info->row_pointers[cinfo.output_scanline]), 1);
    }

    /* Step 7: Finish decompression */
    (void) jpeg_finish_decompress(&cinfo);

    /* Step 8: Release JPEG decompression object & file */
    jpeg_destroy_decompress(&cinfo);
    fclose(infile);

    /* At this point you may want to check to see whether any corrupt-data
    * warnings occurred (test whether jerr.pub.num_warnings is nonzero).
    */

    /* And we're done! */
    return p_info;
}

/*
 * allocate/destroy_frame allocate a frame_struct_t and fill in the
 *  blanks appropriately (including allocating the actual frames), and
 *  then destroy them afterwards.
 */
frame_ptr allocate_frame(int height, int width, int num_channels) {
    int row_stride;		/* physical row width in output buffer */
    int i;
    frame_ptr p_info;		/* Output frame information */

    /* JSAMPLEs per row in output buffer */
    row_stride = width * num_channels;

    /* Basic struct and information */
    if ((p_info = (frame_struct_t*) malloc(sizeof(frame_struct_t))) == NULL) {
        fprintf(stderr, "ERROR: Memory allocation failure\n");
        exit(EXIT_FAILURE);
    }
    p_info->image_height = height;
    p_info->image_width = width;
    p_info->num_channels = num_channels;

    /* Image array and pointers to rows */
    if ((p_info->row_pointers = (JSAMPLE**) malloc(sizeof(JSAMPLE *) * height)) == NULL) {
        fprintf(stderr, "ERROR: Memory allocation failure\n");
        exit(EXIT_FAILURE);
    }
    if ((p_info->image_buffer = (JSAMPLE*) calloc(1, sizeof(JSAMPLE) * row_stride * height)) == NULL) {
        fprintf(stderr, "ERROR: Memory allocation failure\n");
        exit(EXIT_FAILURE);
    }
    for (i=0; i < height; i++)
    	p_info->row_pointers[i] = & (p_info->image_buffer[i * row_stride]);

    /* And send it back! */
    return p_info;
}

void destroy_frame(frame_ptr kill_me) {
    free(kill_me->image_buffer);
    free(kill_me->row_pointers);
    free(kill_me);
}

// --------------------- Project Code -----------------------

// Force function inlining
#ifdef _MSC_VER
    #define forceinline __forceinline
#elif defined(__GNUC__)
    #define forceinline inline __attribute__((__always_inline__))
#elif defined(__CLANG__)
    #if __has_attribute(__always_inline__)
        #define forceinline inline __attribute__((__always_inline__))
    #else
        #define forceinline inline
    #endif
#else
    #define forceinline inline
#endif

#define BYTE_PER_KB 1024
#define BYTE_PER_MB 1048576
#define RUN_TEST 0 // Whether or not to test against uniprocessor
#define JPEG_OUTPUT_QUALITY 75
#define R_GRAYSCALE 0.2126
#define G_GRAYSCALE 0.7152
#define B_GRAYSCALE 0.0722
#define TWO_PI (HIP_PI_F * 2)
#define EIGEN_THRESHOLD 0.01
#define NUM_RUN 10

/**
 * Makes sure the two input frames have the same dimensions
 */
void checkFrameDim(frame_ptr f1, frame_ptr f2) {
    if (
        f1->image_height != f2->image_height ||
        f1->image_width != f2->image_width ||
        f1->num_channels != f2->num_channels) {
        fprintf(stderr, "Dimensions do not match\n");
        exit(EXIT_FAILURE);
    }
}

/**
 * Makes sure values match in the two frames.
 * If there is a difference of 1, ignore because of rounding error.
 */
void checkResults(frame_ptr f1, frame_ptr f2) {
    checkFrameDim(f1, f2);
    int i, j, k;

    for (i = 0; i < f1->image_height; i++){
        for (j = 0; j < f1->image_width; j++){
            for (k = 0; k < f1->num_channels; k++){
                JSAMPLE j1 = f1->row_pointers[i][(f1->num_channels) * j + k];
                JSAMPLE j2 = f2->row_pointers[i][(f2->num_channels) * j + k];
                if (abs(j1 - j2) > 1) {
                    fprintf(stderr, "Values do not match at (%d, %d, %d) \n", i, j, k);
                    fprintf(stderr, "in %d\n", j1);
                    fprintf(stderr, "to %d\n", j2);
                    exit(EXIT_FAILURE);
                }
            }
        }
    }
}

/**
 * Queries the properties of the GPU device with the given device_id,
 * fills it in the given device_prop, and print interesting info
 *
 * @param device_prop The cuda device properties struct
 * @param device_id The id of the gpu device
 */
void query_device(hipDeviceProp_t *device_prop, int device_id) {
    hipGetDeviceProperties(device_prop, device_id);
    printf(
        "******* GPU Device Properties *******\n"
        "Name: %s\n"
        "Compute capabability: %d.%d\n"
        "Clock rate: %dMHz | Memory clock rate: %dMHz\n"
        "Total constant memory: %zu KB | Total global memory: %zu MB\n"
        "Streaming processor (SM) count: %d\n"
        "Shared memory per SM: %zu B | per block: %zu B\n"
        "Registers per SM: %d | per block: %d\n"
        "Max threads per SM: %d | per block: %d\n"
        "Max grid size: %d %d %d\n"
        "Max block dimension: %d %d %d\n"
        "Warp size: %d\n"
        "*************************************\n",
        device_prop->name,
        device_prop->major, device_prop->minor,
        device_prop->clockRate / 1000, device_prop->memoryClockRate / 1000,
        device_prop->totalConstMem / BYTE_PER_KB, device_prop->totalGlobalMem / BYTE_PER_MB,
        device_prop->multiProcessorCount,
        device_prop->sharedMemPerMultiprocessor, device_prop->sharedMemPerBlock,
        device_prop->regsPerMultiprocessor, device_prop->regsPerBlock,
        device_prop->maxThreadsPerMultiProcessor, device_prop->maxThreadsPerBlock,
        device_prop->maxGridSize[0], device_prop->maxGridSize[1], device_prop->maxGridSize[2],
        device_prop->maxThreadsDim[0], device_prop->maxThreadsDim[1], device_prop->maxThreadsDim[2],
        device_prop->warpSize
    );
}

/**
 * Convert HSV to RGB. Saturation is assumed to be 1
 * @param h Hue [0, 360]
 * @param v Value [0, 1].
 * @return A float3 with R, G, B in [0, 1]
 */
inline __host__ __device__ float3 hsv2rgb(float h, float v) {
    h = clamp(h / 360.0f, 0.0f, 1.0f); // Make sure hue is in [0, 1]
    float R = abs(h * 6 - 3) - 1;
    float G = 2 - abs(h * 6 - 2);
    float B = 2 - abs(h * 6 - 4);
    return clamp(make_float3(R, G, B), 0.0f, 1.0f) * v;
}

/**
 * Convert RGB to grayscale (betwene 0 and 1)
 */
inline __host__ __device__ float rgb2gray(unsigned char r, unsigned char g, unsigned char b) {
    return round(R_GRAYSCALE * r + G_GRAYSCALE * g + B_GRAYSCALE * b) / 255.0f;
}

/**
 * Get angle in degrees [0, 360) from flow
 */
inline __host__ __device__ float get_angle(float2 flow) {
    return fmodf((atan2f(flow.y, flow.x) + TWO_PI), TWO_PI) * 180.0f / HIP_PI_F;
}

/**
 * Get magnitude from flow
 */
inline __host__ __device__ float get_magnitude(float2 flow) {
    return sqrt(flow.x * flow.x + flow.y * flow.y);
}

/**
 * Divide then take ceiling
 */
inline __host__ __device__ int divide_up(int a, int b) {
    return (int) ceil(a / (float) b);
}

/**
 * Calculate flow using (A^T A)^{-1} A^T b.
 * @param AtA_00 The upper left entry of A^T A, corresponding to sum of (fx)^2
 * @param AtA_01 The upper right/bottom left entry of A^T A, corresponding to sum of fx * fy
 * @param AtA_11 The bottom right entry of A^T A, corresponding to sum of (fy)^2
 * @param Atb_0 The top entry of A^T b, corresponding to sum of fx * ft
 * @param Atb_1 The bottom entry of A^T b, corresponding to sum of fy * ft
 * @return A float2 containing the flow
 */
inline __host__ __device__ float2 calc_flow_from_matrix(
    float AtA_00, float AtA_01, float AtA_11, float Atb_0, float Atb_1
) {
     // Calculate determinant and make sure it is not 0 in order for the matrix to be invertible
    float det = AtA_00 * AtA_11 - AtA_01 * AtA_01;
    if (det == 0.0f) {
        return make_float2(0.0f, 0.0f);
    }

    // Calculate the eigenvalues of A^T A and make sure they are > threshold
    float trace_half = (AtA_00 + AtA_11) / 2.0f; // Half of the trace of A^T A
    float delta = sqrt(trace_half * trace_half - det);
    float eigen1 = trace_half + delta;
    float eigen2 = trace_half - delta;
    if (eigen1 <= EIGEN_THRESHOLD || eigen2 <= EIGEN_THRESHOLD) {
        return make_float2(0.0f, 0.0f);
    }

    // Calculate flow components
    return make_float2(
        AtA_11 * Atb_0 - AtA_01 * Atb_1,
        -AtA_01 * Atb_0 + AtA_00 * Atb_1
    ) / det;
}

/**
 * Allocate a 2d array of dimension height x width, prefilled with 0
 */
float** alloc_2d_float_array(int height, int width) {
    float **ptr;

    if ((ptr = (float **) malloc(height * sizeof(float *))) == NULL) {
        fprintf(stderr, "ERROR: Memory allocation failure\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < height; i++) {
        if ((ptr[i] = (float *) calloc(1, width * sizeof(float *))) == NULL) {
            fprintf(stderr, "ERROR: Memory allocation failure\n");
            exit(EXIT_FAILURE);
        }
    }
    return ptr;
}

/**
 * Allocate a 1d array of length size, prefilled with 0
 */
float* alloc_1d_float_array(int size) {
    float *ptr;
    if ((ptr = (float *) calloc(1, size * sizeof(float))) == NULL) {
        fprintf(stderr, "ERROR: Memory allocation failure\n");
        exit(EXIT_FAILURE);
    }
    return ptr;
}

/**
 * Free a 2d array
 */
void free_2d_float_array(float **arr, int height) {
    for (int row = 0; row < height; row++) {
        free(arr[row]);
    }
    free(arr);
}

/**
 * Normalize RGB frame and grayscale if necessary
 */
float** get_normalized_2d_float_array(frame_ptr in) {
    int height = in->image_height, width = in->image_width;
    float **out = alloc_2d_float_array(height, width);
    float *cur_out_row;
    JSAMPLE *cur_in_row;
    int row, col;

    if (in->num_channels == 3) { // Grayscale
        for (row = 0; row < height; row++) {
            cur_in_row = in->row_pointers[row];
            cur_out_row = out[row];
            for (col = 0; col < width; col++) {
                cur_out_row[col] = rgb2gray(
                    cur_in_row[col * 3],
                    cur_in_row[col * 3 + 1],
                    cur_in_row[col * 3 + 2]
                );
            }
        }
    } else { // Already grayscaled
        for (row = 0; row < height; row++) {
            cur_in_row = in->row_pointers[row];
            cur_out_row = out[row];
            for (col = 0; col < width; col++) {
                cur_out_row[col] = cur_in_row[col] / 255.0f;
            }
        }
    }

    return out;
}

/**
 * Flatten a heigh x width 2d array into a 1d array
 */
float* flatten_2d_float_array(float **src, int height, int width) {
    float *arr;
    if ((arr = (float *) malloc(height * width * sizeof(float))) == NULL) {
        fprintf(stderr, "ERROR: Memory allocation failure\n");
        exit(EXIT_FAILURE);
    }

    for (int row = 0; row < height; row++) {
        memcpy((void *) (arr + row * width), (void *) src[row], width * sizeof(float));
    }

    return arr;
}

/**
 * Pretty-print a 2d array for debugging purpose
 */
void print_2d_float_array(float **arr, int height, int width) {
    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            printf("%.2f ", arr[row][col]);
        }
        printf("\n");
    }
}

/**
 * Pretty-print a 1d array as a 2d array for debugging purpose
 */
void print_1d_float_array_as_2d(float *arr, int height, int width) {
    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            printf("%.2f ", arr[row * width + col]);
        }
        printf("\n");
    }
}

/**
 * Visualize the flow matrix using HSV and convert to RGB.
 */
void create_flow_visualization(
    frame_ptr out, float *angle, float *mag,
    int height, int width, int s
) {
    float cur_mag, max_mag = -INFINITY, min_mag = INFINITY;
    int row, col;

    // Find min and max magnitude
    for (row = s; row < height - s; row++) {
        for (col = s; col < width - s; col++) {
            cur_mag = mag[row * width + col];
            if (cur_mag > max_mag) {
                max_mag = cur_mag;
            }
            if (cur_mag < min_mag) {
                min_mag = cur_mag;
            }
        }
    }
    if (max_mag <= min_mag) {
        return;
    }

    JSAMPLE *cur_row;
    max_mag -= min_mag;
    for (row = s; row < height - s; row++) {
        cur_row = out->row_pointers[row];

        for (col = s; col < width - s; col++) {
            float3 rgb = hsv2rgb(
                angle[row * width + col], // Angle corresponds to Hue
                clamp((mag[row * width + col] - min_mag) / max_mag, 0.0f, 1.0f) // Magnitude (scaled) corresponds to Value
            );
            cur_row[col * 3] = round(rgb.x * 255);
            cur_row[col * 3 + 1] = round(rgb.y * 255);
            cur_row[col * 3 + 2] = round(rgb.z * 255);
        }
    }
}

/**
 * Calculate optical flow with Lucas-Kanade using CPU from 2 normalized frames
 *
 * @param in1 Frame at time t
 * @param in2 Frame at time t + 1
 * @param window_size An odd positive integer >= 3 for the window size
 * @return A frame for visualizing the optical flow.
 */
void uniprocessor_lucas_kanade(
    float **in1, float **in2, frame_ptr out,
    int height, int width, int window_size
) {
    // Calculate derivatives
    float **fx, **fy, **ft;
    fx = alloc_2d_float_array(height, width);
    fy = alloc_2d_float_array(height, width);
    ft = alloc_2d_float_array(height, width);

    int row, col;
    for (row = 0; row < height; row++) {
        for (col = 0; col < width; col++) {
            if (row > 0 && row < height - 1 && col > 0 && col < width - 1) {
                fx[row][col] =
                    in1[row - 1][col + 1] + in1[row][col + 1] + in1[row + 1][col + 1] -
                    in1[row - 1][col - 1] - in1[row][col - 1] - in1[row + 1][col - 1];
                fy[row][col] =
                    in1[row - 1][col - 1] + in1[row - 1][col] + in1[row - 1][col + 1] -
                    in1[row + 1][col - 1] - in1[row + 1][col] - in1[row + 1][col + 1];
            }
            ft[row][col] = in2[row][col] - in1[row][col];
        }
    }

    // Calculate flows
    int i, j;
    float *angle = alloc_1d_float_array(height * width);
    float *mag = alloc_1d_float_array(height * width);
    float AtA_00, AtA_01, AtA_11, Atb_0, Atb_1;
    float2 flow;
    int s = window_size / 2;

    for (row = s; row < height - s; row++) {
        for (col = s; col < width - s; col++) {
            AtA_00 = AtA_01 = AtA_11 = Atb_0 = Atb_1 = 0.0f;
            for (i = row - s; i <= row + s; i++) {
                for (j = col - s; j <= col + s; j++) {
                    AtA_00 += fx[i][j] * fx[i][j];
                    AtA_11 += fy[i][j] * fy[i][j];
                    AtA_01 += fx[i][j] * fy[i][j];
                    Atb_0 -= fx[i][j] * ft[i][j];
                    Atb_1 -= fy[i][j] * ft[i][j];
                }
            }

            // Calculate flow and convert to polar coordinates
            flow = calc_flow_from_matrix(AtA_00, AtA_01, AtA_11, Atb_0, Atb_1);
            angle[row * width + col] = get_angle(flow);
            mag[row * width + col] = get_magnitude(flow);
        }
    }

    // Create and write to output frame
    create_flow_visualization(out, angle, mag, height, width, s);

    // Clean up
    free_2d_float_array(fx, height);
    free_2d_float_array(fy, height);
    free_2d_float_array(ft, height);
    free(angle);
    free(mag);
}

void run_generic_cuda_kernel(
    float **in1, float **in2, frame_ptr out,
    int height, int width, int window_size,
    dim3 derivative_grid_dim, dim3 derivative_block_dim,
    dim3 flow_grid_dim, dim3 flow_block_dim,
    void (*derivative_kernel_ptr)(float *, float *, float *, float *, float *, int, int),
    void (*normal_lucas_kanade_kernel_ptr)(float *, float *, float *, float *, float *, int, int, int),
    void (*tiled_lucas_kanade_kernel_ptr)(float *, float *, float *, float *, float *, int, int, int, int, int, int),
    bool is_tiled, int out_block_size, size_t shared_mem_size,
    int tile_height, int tile_width, int num_tile
) {
    int s = window_size / 2;
    // Allocate mem
    size_t size = height * width * sizeof(float);
    float *flattened_in1, *flattened_in2, *angle, *mag;
    float *d_in1, *d_in2, *d_fx, *d_fy, *d_ft, *d_angle, *d_mag;
    flattened_in1 = flatten_2d_float_array(in1, height, width);
    flattened_in2 = flatten_2d_float_array(in2, height, width);
    checkCudaErrors(hipMalloc((void **) &d_fx, size));
    checkCudaErrors(hipMalloc((void **) &d_fy, size));
    checkCudaErrors(hipMalloc((void **) &d_ft, size));
    checkCudaErrors(hipMalloc((void **) &d_in1, size));
    checkCudaErrors(hipMalloc((void **) &d_in2, size));
    checkCudaErrors(hipMemcpy(d_in1, flattened_in1, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_in2, flattened_in2, size, hipMemcpyHostToDevice));

    derivative_kernel_ptr<<<derivative_grid_dim, derivative_block_dim>>>(
        d_in1, d_in2, d_fx, d_fy, d_ft, height, width
    );

    angle = alloc_1d_float_array(size);
    mag = alloc_1d_float_array(size);
    checkCudaErrors(hipMalloc((void **) &d_angle, size));
    checkCudaErrors(hipMalloc((void **) &d_mag, size));
    checkCudaErrors(hipMemset((void *) d_angle, 0, size));
    checkCudaErrors(hipMemset((void *) d_mag, 0, size));

    if (is_tiled) {
        tiled_lucas_kanade_kernel_ptr<<<flow_grid_dim, flow_block_dim, shared_mem_size>>>(
            d_fx, d_fy, d_ft, d_angle, d_mag, height, width, s, out_block_size, tile_height, num_tile
        );
    } else {
        normal_lucas_kanade_kernel_ptr<<<flow_grid_dim, flow_block_dim>>>(
            d_fx, d_fy, d_ft, d_angle, d_mag, height, width, s
        );
    }

    checkCudaErrors(hipMemcpy(angle, d_angle, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(mag, d_mag, size, hipMemcpyDeviceToHost));
    create_flow_visualization(out, angle, mag, height, width, s);

    free(flattened_in1);
    free(flattened_in2);
    free(angle);
    free(mag);
    checkCudaErrors(hipFree(d_fx));
    checkCudaErrors(hipFree(d_fy));
    checkCudaErrors(hipFree(d_ft));
    checkCudaErrors(hipFree(d_in1));
    checkCudaErrors(hipFree(d_in2));
    checkCudaErrors(hipFree(d_angle));
    checkCudaErrors(hipFree(d_mag));
}

__global__ void simple_derivative_kernel(
    float *in1, float *in2,
    float *fx, float *fy, float *ft,
    int height, int width
) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < height && col < width) {
        int offset = row * width + col;
        float top_left, top_mid, top_right, mid_left, mid_right, bottom_left, bottom_mid, bottom_right;

        if (row > 0 && row < height - 1 && col > 0 && col < width - 1) {
            top_left = in1[offset - width - 1];
            top_mid = in1[offset - width];
            top_right = in1[offset - width + 1];
            bottom_left = in1[offset + width - 1];
            bottom_mid = in1[offset + width];
            bottom_right = in1[offset + width + 1];
            mid_left = in1[offset - 1];
            mid_right = in1[offset + 1];

            fx[offset] = top_right - top_left + mid_right - mid_left + bottom_right - bottom_left;
            fy[offset] = top_left - bottom_left + top_mid - bottom_mid + top_right - bottom_right;
            ft[offset] = in2[offset] - in1[offset];
        } else {
            fx[offset] = fy[offset] = ft[offset] = 0.0f;
        }
    }
}

__global__ void simple_lucas_kanade_kernel(
    float *fx, float *fy, float *ft, float *angle, float *mag,
    int height, int width, int s
) {
    int row = blockDim.y * blockIdx.y + threadIdx.y + s;
    int col = blockDim.x * blockIdx.x + threadIdx.x + s;
    if (row < height - s && col < width - s) {
        float AtA_00, AtA_01, AtA_11, Atb_0, Atb_1; // Entries of (A^T A)^-1 and A^T b
        float cur_fx, cur_fy, cur_ft;
        AtA_00 = AtA_01 = AtA_11 = Atb_0 = Atb_1 = 0.0f;

        for (int i = row - s; i <= row + s; i++) {
            for (int j = col - s; j <= col + s; j++) {
                cur_fx = fx[i * width + j];
                cur_fy = fy[i * width + j];
                cur_ft = ft[i * width + j];

                AtA_00 += cur_fx * cur_fx;
                AtA_11 += cur_fy * cur_fy;
                AtA_01 += cur_fx * cur_fy;
                Atb_0 -= cur_fx * cur_ft;
                Atb_1 -= cur_fy * cur_ft;
            }
        }

        // Calculate flow and convert to polar coordinates
        float2 flow = calc_flow_from_matrix(AtA_00, AtA_01, AtA_11, Atb_0, Atb_1);
        angle[row * width + col] = get_angle(flow);
        mag[row * width + col] = get_magnitude(flow);
    }
}

void run_simple_kernel(
    float **in1, float **in2, frame_ptr out,
    int height, int width,
    int window_size, int block_size
) {
    int out_width = width - window_size + 1;
    int out_height = height - window_size + 1;
    dim3 block_dim(block_size, block_size, 1);
    dim3 derivative_grid_dim(divide_up(width, block_size), divide_up(height, block_size), 1);
    dim3 flow_grid_dim(divide_up(out_width, block_size), divide_up(out_height, block_size), 1);

    run_generic_cuda_kernel(
        in1, in2, out, height, width, window_size,
        derivative_grid_dim, block_dim, flow_grid_dim, block_dim,
        simple_derivative_kernel, simple_lucas_kanade_kernel, NULL, false,
        0, 0, 0, 0, 0
    );
}

// ----- Tiled kernel -----

/**
 * blockDim.x is the tile's width, and its square is the number of floats
 */
__global__ void tiled_lucas_kanade_kernel(
    float *fx, float *fy, float *ft, float *angle, float *mag,
    int height, int width, int s, int out_block_size, int tile_height, int num_tile
) {
    // Get column for loading data
    int cur_col = out_block_size * blockIdx.x + threadIdx.x;
    if (cur_col >= width) {
        return;
    }

    // Load data into shared mem tile by tile
    extern __shared__ float shared_mem[]; // Contains fx, fy, ft
    int offset;
    int cur_row = out_block_size * blockIdx.y - tile_height + threadIdx.y;
    for (int k = 0; k < num_tile; k++) {
        cur_row += tile_height;
        offset = 3 * ((k * tile_height + threadIdx.y) * blockDim.x + threadIdx.x);
        if (cur_row < height && (k * tile_height + threadIdx.y) < blockDim.x) {
            shared_mem[offset] = fx[cur_row * width + cur_col];
            shared_mem[offset + 1] = fy[cur_row * width + cur_col];
            shared_mem[offset + 2] = ft[cur_row * width + cur_col];
        }
    }

    __syncthreads(); // Wait for memory loading

    // Get rid of unnecessary threads based on column
    if (threadIdx.x < s || threadIdx.x >= blockDim.x - s || cur_col >= width - s) {
        return;
    }

    // Start calculating flow tile by tile
    float AtA_00, AtA_01, AtA_11, Atb_0, Atb_1; // Entries of (A^T A)^-1 and A^T b
    float cur_fx, cur_fy, cur_ft;
    int cur_block_row = threadIdx.y - tile_height;
    cur_row = out_block_size * blockIdx.y - tile_height + threadIdx.y; // reset row

    for (int k = 0; k < num_tile; k++) {
        cur_row += tile_height;
        cur_block_row += tile_height;

        if (cur_block_row >= s && cur_block_row < blockDim.x - s && cur_row < height - s) {
            AtA_00 = AtA_01 = AtA_11 = Atb_0 = Atb_1 = 0.0f;
            for (int row = cur_block_row - s; row <= cur_block_row + s; row++) {
                for (int col = threadIdx.x - s; col <= threadIdx.x + s; col++) {
                    offset = 3 * (row * blockDim.x + col);
                    cur_fx = shared_mem[offset];
                    cur_fy = shared_mem[offset + 1];
                    cur_ft = shared_mem[offset + 2];

                    AtA_00 += cur_fx * cur_fx;
                    AtA_11 += cur_fy * cur_fy;
                    AtA_01 += cur_fx * cur_fy;
                    Atb_0 -= cur_fx * cur_ft;
                    Atb_1 -= cur_fy * cur_ft;
                }
            }

            // Calculate flow and convert to polar coordinates
            float2 flow = calc_flow_from_matrix(AtA_00, AtA_01, AtA_11, Atb_0, Atb_1);
            angle[cur_row * width + cur_col] = get_angle(flow);
            mag[cur_row * width + cur_col] = get_magnitude(flow);
        }
    }
}

void run_tiled_kernel(
    float **in1, float **in2, frame_ptr out,
    int height, int width,
    int window_size, int out_block_size, int max_threads_per_block
) {
    // Grid and block dim for the derivative kernel
    dim3 derivative_block_dim(out_block_size, out_block_size, 1);
    dim3 derivative_grid_dim(divide_up(width, out_block_size), divide_up(height, out_block_size), 1);

    // Grid, block dim, tile dim, and shared mem size for the tiled flow kernel
    int in_block_size = out_block_size + window_size - 1;
    int tile_height = min(in_block_size, max_threads_per_block / in_block_size);
    int out_width = width - window_size + 1;
    int out_height = height - window_size + 1;
    dim3 flow_block_dim(in_block_size, tile_height, 1);
    dim3 flow_grid_dim(divide_up(out_width, out_block_size), divide_up(out_height, out_block_size), 1);
    size_t shared_mem_size = 3 * in_block_size * in_block_size * sizeof(float); // reserve shared mem for fx, fy, ft
    int num_tile = divide_up(in_block_size, tile_height);

    run_generic_cuda_kernel(
        in1, in2, out, height, width, window_size,
        derivative_grid_dim, derivative_block_dim, flow_grid_dim, flow_block_dim,
        simple_derivative_kernel, NULL, tiled_lucas_kanade_kernel, true,
        out_block_size, shared_mem_size, tile_height, in_block_size, num_tile
    );
}

size_t calc_max_window_size(hipDeviceProp_t *device_prop, int out_block_size) {
    // Maximum number of floats per shared mem array (fx, fy, ft)
    int max_float_num = device_prop->sharedMemPerBlock / sizeof(float) / 3;
    int max_window_size = ((int) floor(sqrt(max_float_num))) - out_block_size + 1;
    return max_window_size % 2 == 1 ? max_window_size : max_window_size - 1;
}

/**
 * Host main routine
 */
int main(int argc, char **argv) {
    if (argc != 5) {
        fprintf(stderr,
            "Usage: lucas_kanade <WINDOW_SIZE>"
            " <PATH_TO_FRAME_1> <PATH_TO_FRAME_2> <PATH_TO_FLOW_OUTPUT>\n");
        exit(EXIT_FAILURE);
    }

    // Get inputs
    frame_ptr raw_in1 = read_JPEG_file(argv[2]);
    frame_ptr raw_in2 = read_JPEG_file(argv[3]);
    checkFrameDim(raw_in1, raw_in2);

    int window_size = atoi(argv[1]);
    if (window_size < 3 || window_size % 2 != 1) {
        fprintf(stderr, "Window size must be an odd integer >= 3\n");
        exit(EXIT_FAILURE);
    }

    // Create 2d array of normalized image pixel between [0, 1]
    int height = raw_in1->image_height;
    int width = raw_in2->image_width;
    float **in1 = get_normalized_2d_float_array(raw_in1);
    float **in2 = get_normalized_2d_float_array(raw_in2);

    // Get max block size from GPU's props
    int device_id = gpuGetMaxGflopsDeviceId();
    checkCudaErrors(hipSetDevice(device_id));
    hipDeviceProp_t device_prop;
    query_device(&device_prop, device_id);
    int max_block_size = (int) floor(sqrt(device_prop.maxThreadsPerBlock));

    // Check window size for tiling
    int max_window_size = calc_max_window_size(&device_prop, max_block_size);
    if (max_window_size < window_size) {
        fprintf(stderr, "Window size must be at most %d\n", max_window_size);
        exit(EXIT_FAILURE);
    }

    // Allocate output frames
    frame_ptr out_gpu = allocate_frame(height, width, 3);
    frame_ptr out_cpu = allocate_frame(height, width, 3);

    // Run CPU version
    uniprocessor_lucas_kanade(in1, in2, out_cpu, height, width, window_size);

    // Run GPU version several times for profiler while also testing against CPU version
    for (int i = 0; i < NUM_RUN; i++) {
        run_simple_kernel(in1, in2, out_gpu, height, width, window_size, max_block_size);
        checkResults(out_gpu, out_cpu);
        run_tiled_kernel(in1, in2, out_gpu, height, width, window_size, max_block_size, device_prop.maxThreadsPerBlock);
        checkResults(out_gpu, out_cpu);
    }

    // Write out the visualization and clean up
    write_JPEG_file(argv[4], out_gpu, JPEG_OUTPUT_QUALITY);
    destroy_frame(raw_in1);
    destroy_frame(raw_in2);
    free_2d_float_array(in1, height);
    free_2d_float_array(in2, height);
    destroy_frame(out_gpu);
    destroy_frame(out_cpu);
    return 0;
}
